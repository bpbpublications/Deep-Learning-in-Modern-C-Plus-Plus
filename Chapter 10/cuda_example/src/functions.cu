#include <iostream>

#include "functions.hpp"

void printCudaVersion()
{
    std::cout
        << "__CUDACC_VER_MAJOR__: " << __CUDACC_VER_MAJOR__ << "\n"
        << "__CUDACC_VER_MINOR__" << __CUDACC_VER_MINOR__ << "\n"
        << "__CUDACC_VER_BUILD__: " << __CUDACC_VER_BUILD__ << "\n\n";

    int runtime_ver;
    hipRuntimeGetVersion(&runtime_ver);
    std::cout << "CUDA Runtime version: " << runtime_ver << "\n";

    int driver_ver;
    hipDriverGetVersion(&driver_ver);
    std::cout << "CUDA Driver version: " << driver_ver << "\n";
}

Tensor<float, 2> run_gpu_contraction(const Tensor<float, 2> &A, const Tensor<float, 2> &B)
{
    const int M = A.dimension(0);
    const int K = A.dimension(1); // for production, assert K == B.dimension(0)
    const int N = B.dimension(1);

    // allocating data on GPU

    std::size_t A_bytes = M * K * sizeof(float);
    std::size_t B_bytes = K * N * sizeof(float);

    float *A_data;
    float *B_data;

    gpuMalloc((void **)(&A_data), A_bytes);
    gpuMalloc((void **)(&B_data), B_bytes);

    gpuMemcpy(A_data, A.data(), A_bytes, gpuMemcpyHostToDevice);
    gpuMemcpy(B_data, B.data(), B_bytes, gpuMemcpyHostToDevice);

    auto M_array = Eigen::array<int, 2>{M, K};
    auto K_array = Eigen::array<int, 2>{K, N};
    Eigen::TensorMap<Eigen::Tensor<float, 2>> A_on_GPU(A_data, M_array);
    Eigen::TensorMap<Eigen::Tensor<float, 2>> B_on_GPU(B_data, K_array);

    std::size_t result_bytes = M * N * sizeof(float);
    float *result_data;
    gpuMalloc((void **)(&result_data), result_bytes);
    auto n_array = Eigen::array<int, 2>{M, N};
    Eigen::TensorMap<Eigen::Tensor<float, 2>> gpu_result(result_data, n_array);

    // running contraction on GPU
    Eigen::GpuStreamDevice stream;
    Eigen::GpuDevice gpu_device(&stream);
    Eigen::array<Eigen::IndexPair<int>, 1> dims = {Eigen::IndexPair<int>(1, 0)};
    gpu_result.device(gpu_device) = A_on_GPU.contract(B_on_GPU, dims);

    // copying the result data to CPU
    Tensor<float, 2> result(M, N);
    gpuMemcpy(result.data(), result_data, result_bytes, gpuMemcpyDeviceToHost);

    // freed GPU memory
    gpuFree((void *)A_data);
    gpuFree((void *)B_data);
    gpuFree((void *)result_data);

    return result;
}
